#include "hip/hip_runtime.h"
#include "multi_block_per_sample_preprocess.cuh"
#include <hipcub/hipcub.hpp>

namespace mbs {

void StringSplitToInt(const std::vector<std::string> &inputs,
                      const int max_concurrent_blocks, const int tile_size,
                      std::vector<int> &indices,
                      std::vector<int> &segment_offsets,
                      std::vector<int> &workload_mapping, int &num_tiles,
                      int &num_blocks, int &num_inputs, int &num_segments) {
  const int bs = inputs.size();
  indices = std::vector<int>();
  segment_offsets = std::vector<int>(bs + 1, 0);
  workload_mapping = std::vector<int>();

  num_tiles = 0;
  for (int seg_id = 0; seg_id < bs; ++seg_id) {
    int index = 0;
    int count = 0;
    for (char ch : inputs[seg_id]) {
      if (ch < '0' || ch > '9') {
        indices.push_back(index);
        ++count;
        index = 0;
      } else {
        index = index * 10 + (ch - '0');
      }
    }
    if (inputs[seg_id].size() > 0) {
      indices.push_back(index);
      ++count;
    }

    segment_offsets[seg_id + 1] = segment_offsets[seg_id] + count;
    int tiles_for_sample = hipcub::DivideAndRoundUp(count, tile_size);
    num_tiles += tiles_for_sample;
    for (int sample_tile_idx = 0; sample_tile_idx < tiles_for_sample;
         ++sample_tile_idx) {
      workload_mapping.push_back(seg_id);
      workload_mapping.push_back(sample_tile_idx);
    }
  }

  num_inputs = indices.size();
  num_segments = bs;
  num_blocks = std::min(num_tiles, max_concurrent_blocks);
}

} // namespace mbs